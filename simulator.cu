#include "hip/hip_runtime.h"
#include "simulator.h"
#include "kernel.cu"
#include <vector>
#include <cstdio>
#include <iostream>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    exit(EXIT_FAILURE);}} while(0)

#define MALLOC(x, y) CUDA_CALL(hipMallocManaged(x, y))

const int block_size = 512;
const int decoders = 100;
const int blocks = decoders;
const float SNR = 2;

void fillInput(CodeInfo**, Edge**, Edge**);

int main()
{
    CodeInfo* codeInfo;
    Edge* edgesFromVariable;
    Edge* edgesFromCheck;
    fillInput(&codeInfo, &edgesFromVariable, &edgesFromCheck);
    float sigma2 = pow(10.0, -SNR / 10);

    float* probP;
    float* probQ;
    float* probR;
    MALLOC(&probP, decoders * codeInfo->varNodes * sizeof(float));
    MALLOC(&probQ, decoders * codeInfo->totalEdges * sizeof(float));
    MALLOC(&probR, decoders * codeInfo->totalEdges * sizeof(float));

    float* noisedVector;
    int noisedVectorSize = decoders * codeInfo->varNodes;
    MALLOC(&noisedVector, noisedVectorSize);
    // adding noise
    for (int i = 0; i < noisedVectorSize; i++)
    {
        noisedVector[i] = -1;
    }

    // Kernel execution
    decodeAWGN<<<blocks, block_size>>>(
            codeInfo,
            edgesFromVariable,
            edgesFromCheck,
            probP,
            probQ,
            probR,
            sigma2,
            noisedVector);
    //hipMemcpy(berOut, berOut_obj, sizeof(float)
    CUDA_CALL(hipDeviceSynchronize());
}

void fillInput(
        CodeInfo** codeInfo, Edge** edgesFromVariable, Edge** edgesFromCheck)
{
    MALLOC(codeInfo, sizeof(CodeInfo));
    freopen("matrix.txt", "r", stdin);
    int n, k;
    std::cin >> k >> n;
    (*codeInfo)->checkNodes = k;
    (*codeInfo)->varNodes = n;
    std::vector<std::vector<int>> h(k, std::vector<int>(n));
    for (int i = 0; i < k; i++)
    {
        for (int j = 0; j < n; j++)
        {
            std::cin >> h[i][j];
            if (h[i][j])
                (*codeInfo)->totalEdges++;
        }
    }

    MALLOC(edgesFromVariable, (*codeInfo)->totalEdges * sizeof(Edge));
    MALLOC(edgesFromCheck, (*codeInfo)->totalEdges * sizeof(Edge));

    int currentEdge = 0;
    std::vector<std::vector<int>> e(k, std::vector<int>(n));
    for (int j = 0; j < n; j++)
    {
        int connectedToNode = 0;
        int absoluteStartIndex = currentEdge;
        for (int i = k - 1; i >= 0; i--)
        {
            if (h[i][j])
            {
                e[i][j] = currentEdge++;
                Edge& edge = (*edgesFromVariable)[e[i][j]];
                edge.index = e[i][j];
                edge.vn = j;
                edge.cn = i;
                edge.absoluteStartIndex = absoluteStartIndex;
                edge.relativeIndexFromNode = connectedToNode;
                connectedToNode++;
            }
        }
        for (int i = k - 1; i >= 0; i--)
        {
            if (h[i][j])
            {
                Edge& edge = (*edgesFromVariable)[e[i][j]];
                edge.edgesConnectedToNode = connectedToNode;
                /*std::cout << edge.index << " " << edge.vn << " " << edge.cn
                    << " " << edge.edgesConnectedToNode
                    << " " << edge.absoluteStartIndex
                    << " " << edge.relativeIndexFromNode << std::endl;*/
            }
        }
    }

    //std::cout << "Table II" << std::endl;
    currentEdge = 0;
    for (int i = 0; i < k; i++)
    {
        int connectedToNode = 0;
        int absoluteStartIndex = currentEdge;
        for (int j = 0; j < n; j++)
        {
            if (h[i][j])
            {
                currentEdge++;
                Edge& edge = (*edgesFromCheck)[e[i][j]];
                edge.index = e[i][j];
                edge.vn = j;
                edge.cn = i;
                edge.absoluteStartIndex = absoluteStartIndex;
                edge.relativeIndexFromNode = connectedToNode;
                connectedToNode++;
            }
        }
        for (int j = 0; j < n; j++)
        {
            if (h[i][j])
            {
                Edge& edge = (*edgesFromCheck)[e[i][j]];
                edge.edgesConnectedToNode = connectedToNode;
                /*std::cout << edge.index << " " << edge.vn << " " << edge.cn
                    << " " << edge.edgesConnectedToNode
                    << " " << edge.absoluteStartIndex
                    << " " << edge.relativeIndexFromNode << std::endl;*/
            }
        }
    }
}
